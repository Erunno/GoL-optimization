#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_hello.cuh"

// Simple CUDA kernel
__global__ void hello_kernel() {
    // Empty kernel just to verify CUDA compilation
    printf("Hello from CUDA kernel!\n");
}

// Function to be called from main
void cuda_hello() {
    // Launch a simple kernel
    hello_kernel<<<1, 1>>>();
    
    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
    
    // Print CUDA information
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    printf("Hello from CUDA! Found %d CUDA device(s)\n", deviceCount);
    
    // Print device information for each device
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        printf("  Device %d: %s\n", i, deviceProp.name);
        printf("    Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    }
}
